#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <assert.h>
#include <stdio.h>

#define ENABLE_NVIDIA_OPTIMIZATIONS

#ifdef ENABLE_NVIDIA_OPTIMIZATIONS
namespace nvidia {

////////////////////////////////////////////////////////////////////////////////////////////////////

constexpr int THREADS_PER_WARP = 32;


constexpr int LOW_OCCUPANCY_THRESHOLD = 42; // TODO: Make it HW specific (like 1/2 SMs).

static inline __device__ __host__ int div_up(int m, int n) {
  return (m + n-1) / n;
}

static inline __device__ __host__ int round_up(int m, int n) {
  return div_up(m, n) * n;
}

template< typename T >
struct Lmha_params {

  // The output buffer. Dimensions [B, H, L, M].
  T *out;

  // The input Qs. Dimensions [B, H, L, E].
  const T *q;
  // The input Ks. Dimensions [B, H, L, E].
  const T *k;
  // The input Vs. Dimensions [B, H, L, M].
  const T *v;
  // The input gammas. Dimensions[H,],
  const T *gamma;

  // The different dimensions.
  int B, L, H, E, M;

  // The strides for the different tensors.
  int q_stride_B, q_stride_H, q_stride_L;
  int k_stride_B, k_stride_H, k_stride_L;
  int v_stride_B, v_stride_H, v_stride_L;
  int o_stride_B, o_stride_H, o_stride_L;
};

////////////////////////////////////////////////////////////////////////////////////////////////////

template< int E, int WARPS, int COLS_PER_THREAD = 4 >
__global__ __launch_bounds__(WARPS * THREADS_PER_WARP)
void lmha_low_occupancy_kernel(Lmha_params<float> params) {

  // The number of threads per block.
  constexpr int THREADS_PER_BLOCK = WARPS * THREADS_PER_WARP;
  // The number of rows per thread.
  constexpr int ROWS_PER_THREAD = E / THREADS_PER_WARP;
  // The number of steps per iteration.
  constexpr int COLS_PER_ITER = WARPS * COLS_PER_THREAD;

  // Make sure E is a multiple of the warp size.
  static_assert(E % THREADS_PER_WARP == 0, "");

  // Shared memory to store V/O.
  __shared__ float smem_v[COLS_PER_ITER], smem_o[COLS_PER_ITER];
  // Shared memory buffer to performance the reductions.
  __shared__ float smem_reds[E * WARPS]; 

  // The sequence processed by that block.
  const int bi = blockIdx.z;
  // The head processed by that block.
  const int hi = blockIdx.y;
  // The hidden cell in the V/output buffers.
  const int vi = blockIdx.x;

  // The linear index of the thread.
  const int tidx = threadIdx.x;

  // Decompose the block in warp/lane.
  const int warp = tidx / THREADS_PER_WARP;
  const int lane = tidx % THREADS_PER_WARP;

  // The base offset loaded by the thread in Q and K.
  int offset_q = bi*params.q_stride_B + hi*params.q_stride_H + lane; 
  int offset_k = bi*params.k_stride_B + hi*params.k_stride_H + lane;

  // Position the warp at the beginning of the proper timestep.
  offset_q += warp*COLS_PER_THREAD*params.q_stride_L;
  offset_k += warp*COLS_PER_THREAD*params.k_stride_L;


  // Determine the base pointers for Q and K.
  const float *ptr_q = &params.q[offset_q];
  const float *ptr_k = &params.k[offset_k];

  // Is a given row valid?
  int valid_qk[ROWS_PER_THREAD];
  #pragma unroll
  for( int ii = 0; ii < ROWS_PER_THREAD; ++ii ) {
    valid_qk[ii] = lane + ii*THREADS_PER_WARP < params.E;
  }

  // The offset to the position loaded by the thread in V.
  int offset_v = bi*params.v_stride_B + hi*params.v_stride_H + vi;
  int offset_o = bi*params.o_stride_B + hi*params.o_stride_H + vi;

  // We load/store a strided matrix of COLS_PER_ITER x OUTPUTS_PER_BLOCK.

  offset_v += tidx*params.v_stride_L;
  offset_o += tidx*params.o_stride_L;

  // Determine the base pointer for V.
  const float *ptr_v = &params.v[offset_v];
  // The output pointer. 
  float *ptr_o = &params.out[offset_o];

  // The running KVs.
  float running_kv[ROWS_PER_THREAD];
  #pragma unroll
  for( int ri = 0; ri < ROWS_PER_THREAD; ++ri ) {
    running_kv[ri] = 0.f;
  }

  // Iterate over the timesteps. TODO: Use params.loop_count!!!
  for( int iter = 0; iter < params.L; iter += COLS_PER_ITER ) {

    // Each thread loads a matrix of elements.
    float q[ROWS_PER_THREAD][COLS_PER_THREAD], k[ROWS_PER_THREAD][COLS_PER_THREAD];

    // Trigger the memory loads for Q and K.
    #pragma unroll
    for( int ci = 0; ci < COLS_PER_THREAD; ++ci ) {
      #pragma unroll
      for( int ri = 0; ri < ROWS_PER_THREAD; ++ri ) {

        // For Q/K, each warp loads from various timesteps. 
        int ti = iter + warp*COLS_PER_THREAD;

        // Is it a valid access?
        int valid;
        valid = valid_qk[ri] && ti + ci < params.L;
        

        // The extra offset to add.
        offset_q = ri*THREADS_PER_WARP + ci*params.q_stride_L;
        offset_k = ri*THREADS_PER_WARP + ci*params.k_stride_L;
    

        // Load Q/K if they are valid.
        q[ri][ci] = valid ? ptr_q[offset_q] : 0.f;
        k[ri][ci] = valid ? ptr_k[offset_k] : 0.f;
      }
    }

    // For the V tensor, we assign contiguous thread to different loads. So, ti is different.
    int ti = iter + tidx;

    // Is it a valid access?
    int valid_vo = tidx < COLS_PER_ITER;
    valid_vo &= ti < params.L;
    

    // Trigger the loads for V. 
    float ldg_v = valid_vo ? *ptr_v : 0.f;

    // Move the load pointers.
    ptr_q += COLS_PER_ITER*params.q_stride_L;
    ptr_k += COLS_PER_ITER*params.k_stride_L;
    ptr_v += COLS_PER_ITER*params.v_stride_L;
    

    // Store to shared memory.
    if( tidx < COLS_PER_ITER ) {
      smem_v[tidx] = ldg_v;
    }

    // Make sure V is in shared memory.
    __syncthreads();

    // Read V from shared memory.
    float v[COLS_PER_THREAD];
    #pragma unroll
    for( int ci = 0; ci < COLS_PER_THREAD; ++ci ) {
      v[ci] = smem_v[warp*COLS_PER_THREAD + ci];
    }

    // Each thread computes local K*V products.
    float kv[ROWS_PER_THREAD][COLS_PER_THREAD];
    #pragma unroll
    for( int ri = 0; ri < ROWS_PER_THREAD; ++ri ) {
      #pragma unroll
      for( int ci = 0; ci < COLS_PER_THREAD; ++ci ) {
        kv[ri][ci] = 0.f;
      }
    }

    // Update the K*V^T product.
    #pragma unroll
    for( int ci = 0; ci < COLS_PER_THREAD; ++ci ) {
      #pragma unroll
      for( int ri = 0; ri < ROWS_PER_THREAD; ++ri ) {
        kv[ri][ci] += k[ri][ci] * v[ci];
      }
    }

    // We must perform the prefix sums within the thread-block. Start with the thread.
    #pragma unroll
    for( int ri = 0; ri < ROWS_PER_THREAD; ++ri ) {
      #pragma unroll
      for( int ci = 1; ci < COLS_PER_THREAD; ++ci ) {
        kv[ri][ci] += kv[ri][ci-1];
      }
    }

    // Store the partial sums to shared memory. Unless we have no inter-warp reduction to perform.
    #pragma unroll
    for( int ri = 0; ri < ROWS_PER_THREAD; ++ri ) {
      smem_reds[warp*E + ri*THREADS_PER_WARP + lane] = kv[ri][COLS_PER_THREAD-1];
    }

    // Make sure the data is in shared memory.
    __syncthreads();

    // Each thread deals with one or more column(s) of the matrix.
    constexpr int SUMS_PER_THREAD = (E + THREADS_PER_BLOCK-1) / THREADS_PER_BLOCK;
    #pragma unroll
    for( int ii = 0, idx = tidx; ii < SUMS_PER_THREAD; ++ii, idx += THREADS_PER_BLOCK ) {
      if( idx < E ) {
        float sum = smem_reds[idx];
        #pragma unroll
        for( int jj = 1; jj < WARPS; ++jj ) {
          smem_reds[idx + jj*E] = sum += smem_reds[idx + jj*E];
        }
      }
    }

    // Make sure the reductions are stored in shared memory.
    __syncthreads();

    // Each thread updates his partial products.
    #pragma unroll
    for( int ri = 0; ri < ROWS_PER_THREAD; ++ri ) {
      float sum = running_kv[ri];
      if( warp > 0 ) {
        sum += smem_reds[(warp-1)*E + lane + ri*THREADS_PER_WARP];
      }
      #pragma unroll
      for( int ci = 0; ci < COLS_PER_THREAD; ++ci ) {
        kv[ri][ci] += sum;
      }
    }

    // Compute the partial output values for that thread.
    float sum[COLS_PER_THREAD];
    #pragma unroll
    for( int ci = 0; ci < COLS_PER_THREAD; ++ci ) {
      sum[ci] = q[0][ci] * kv[0][ci];
      #pragma unroll
      for( int ri = 1; ri < ROWS_PER_THREAD; ++ri ) {
        sum[ci] += q[ri][ci] * kv[ri][ci];
      }
    }

    // Run the parallel reductions inside the warp.
    #pragma unroll
    for( int mask = THREADS_PER_WARP / 2; mask >= 1; mask /= 2 ) {
      #pragma unroll
      for( int ci = 0; ci < COLS_PER_THREAD; ++ci ) {
        sum[ci] += __shfl_xor_sync(uint32_t(-1), sum[ci], mask);
      }
    }

    // Store the final output to shared memory.
    if( lane == 0 ) {
      #pragma unroll
      for( int ci = 0; ci < COLS_PER_THREAD; ++ci ) {
        smem_o[warp*COLS_PER_THREAD + ci] = sum[ci];
      }
    }

    // Make sure the data is in shared memory.
    __syncthreads();

    // Store the output.
    if( valid_vo ) {
      *ptr_o = smem_o[tidx];
    }

    // Each thread updates his running kv.
    #pragma unroll
    for( int ri = 0; ri < ROWS_PER_THREAD; ++ri ) {
      running_kv[ri] += smem_reds[(WARPS-1)*E + lane + ri*THREADS_PER_WARP];
    }

    // Move to next location.
    ptr_o += COLS_PER_ITER*params.o_stride_L;
    
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template< int E, int WARPS >
int lmha_low_occupancy_(const Lmha_params<float> &params) {

  // Make sure we are not going to launch an invalid grid.
  if( params.H > 65535 || params.B > 65535 ) {
    return 1;
  }

  // Prepare the grid and trigger the CUDA kernel.
  dim3 grid;
  grid.x = params.M;
  grid.y = params.H;
  grid.z = params.B;
  lmha_low_occupancy_kernel<E, WARPS><<<grid, WARPS*THREADS_PER_WARP>>>(params);
  return 0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template< int E>
int lmha_low_occupancy_(const Lmha_params<float> &params, int blocks) {
         if( params.M * blocks >= 8*LOW_OCCUPANCY_THRESHOLD ) {
    return lmha_low_occupancy_<E,  4>(params);
  } else if( params.M * blocks >= 4*LOW_OCCUPANCY_THRESHOLD ) {
    return lmha_low_occupancy_<E,  8>(params);
  } else {
    return lmha_low_occupancy_<E, 16>(params);
  }
  return 1;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template< int E, typename Params >
static inline __device__ __host__ int smem_buffer_elts_(const Params &params) {
  int M = round_up(params.M, 4);
  return 2*E + 2*M;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template< int E, int THREADS_PER_HEAD>
__global__ 
void lmha_kernel(Lmha_params<float> params) {

  // Make sure E is a multiple of 4.
  static_assert(E % 4 == 0, "");

  // The amount of shared memory per buffer (2 buffers for double-buffering).
  const int smem_buffer_elts = smem_buffer_elts_<E>(params);
  // The M dimension for shared memory.
  const int M = round_up(params.M, 4);

  // Shared memory to store Q, K and V. Size is 2*smem_buffer_elts.
  extern __shared__ float smem_[];

  // The various shared memory buffers.
  // 存储Q，K，V，O的首地址，因为V一行有M个元素，所以sem_o首地址从2*E+M开始，因为要存储Q，K，V，O的首地址，所以seme的大小为2E+2M
  float *smem_q = &smem_[0*E];
  float *smem_k = &smem_[1*E];
  float *smem_v = &smem_[2*E];
  float *smem_o = &smem_[2*E + M];

  // The index of the shared memory buffer (for double-buffering).
  int smem_curr = 0;

  // The sequence processed by that block.
  const int bi = blockIdx.y;
  // The head processed by that block.
  const int hi = blockIdx.x;

  // The linear index of the thread.
  const int tidx = threadIdx.x;

  // The offset to the position loaded by the thread in Q.
  int offset_q = bi*params.q_stride_B + hi*params.q_stride_H + tidx;
  // The offset to the position loaded by the thread in K.
  int offset_k = bi*params.k_stride_B + hi*params.k_stride_H + tidx;


  // Determine the base pointers for Q and K.
  const float *ptr_q = &params.q[offset_q];
  const float *ptr_k = &params.k[offset_k];

  // The offset to the position loaded by the thread in V and O.
  int offset_v = bi*params.v_stride_B + hi*params.v_stride_H + tidx;
  int offset_o = bi*params.o_stride_B + hi*params.o_stride_H + tidx;


  // Determine the base pointers for V.
  const float *ptr_v = &params.v[offset_v];

  // Determine the base pointer for gamma
  float gamma = params.gamma[hi];

  // Is it an active Q/K thread?
  const int active_qk = tidx < params.E;

  // Trigger the memory loads for Q and K.
  float ldg_q = 0.f, ldg_k = 0.f;
  if( active_qk ) {
    ldg_q = *ptr_q;
    ldg_k = *ptr_k;
  }

  // Is it an active V thread?
  const int active_v = tidx < params.M;

  // Trigger the memory loads for V. 
  float ldg_v = 0.f;
  if( active_v ) {
    ldg_v = *ptr_v;
  }

  // Move the load pointers.
  ptr_q += params.q_stride_L;
  ptr_k += params.k_stride_L;
  ptr_v += params.v_stride_L;

  // The number of FLOAT4s per head.
  constexpr int FLOAT4s_PER_HEAD = E / 4;
  // The number of FLOAT4s per thread.
  constexpr int FLOAT4s_PER_THREAD = FLOAT4s_PER_HEAD / THREADS_PER_HEAD;

  // The storage for the K*V^T values.
  float4 kv[FLOAT4s_PER_THREAD]; 
  #pragma unroll
  for( int ii = 0; ii < FLOAT4s_PER_THREAD; ++ii ) {
    kv[ii] = make_float4(0.f, 0.f, 0.f, 0.f);
  }

  // The output pointer.
  float *out_ptr = &params.out[offset_o];

  // Store to shared memory Q and K.
  if( tidx < E ) { 
    smem_q[smem_curr*smem_buffer_elts + tidx] = ldg_q; 
    smem_k[smem_curr*smem_buffer_elts + tidx] = ldg_k; 
  }

  // Store to shared memory V. All threads store valid values.
  if( tidx < M ) {
    smem_v[smem_curr*smem_buffer_elts + tidx] = ldg_v;
  }

  // The position of the thread in the V dimension.
  int vo = tidx / THREADS_PER_HEAD;
  int vi = tidx % THREADS_PER_HEAD;

  // Iterate over the timesteps.
  for( int ti = 0; ti < params.L; ++ti ) {

    // Is it the last iteration?
    int is_last = ti == params.L - 1;

    // Trigger the next loads for Q and K.
    if( !is_last && active_qk ) {
      ldg_q = *ptr_q;
      ldg_k = *ptr_k;
    }

    // Trigger the next loads for V.
    if( !is_last && active_v ) {
      ldg_v = *ptr_v;
    }

    // Move the load pointers.
    ptr_q += params.q_stride_L;
    ptr_k += params.k_stride_L;
    ptr_v += params.v_stride_L;

    // Make sure the data is in shared memory.
    __syncthreads();

    // Each thread loads 4 values from K.
    float4 k[FLOAT4s_PER_THREAD];
    #pragma unroll
    for( int ii = 0; ii < FLOAT4s_PER_THREAD; ++ii ) {
      int ki = tidx % THREADS_PER_HEAD * 4 + ii * THREADS_PER_HEAD * 4;
      k[ii] = *reinterpret_cast<const float4*>(&smem_k[smem_curr*smem_buffer_elts + ki]);
    }

    // Each thread loads a single V value.
    float v = 0.f;
    if( vo < params.M ) {
      v = *reinterpret_cast<const float *>(&smem_v[smem_curr*smem_buffer_elts + vo]);
    }

    // Update the K*V^T product.
    #pragma unroll
    for( int ii = 0; ii < FLOAT4s_PER_THREAD; ++ii ) {
      kv[ii].x += k[ii].x * v;
      kv[ii].y += k[ii].y * v;
      kv[ii].z += k[ii].z * v;
      kv[ii].w += k[ii].w * v;
    }

    // Load the Q values from shared memory.
    float4 q[FLOAT4s_PER_THREAD]; 
    #pragma unroll
    for( int ii = 0; ii < FLOAT4s_PER_THREAD; ++ii ) {
      int qi = tidx % THREADS_PER_HEAD * 4 + ii * THREADS_PER_HEAD * 4;
      q[ii] = *reinterpret_cast<const float4*>(&smem_q[smem_curr*smem_buffer_elts + qi]);
    }

    // Compute the partial output value for that thread.
    float sum = 0.f;
    #pragma unroll
    for( int ii = 0; ii < FLOAT4s_PER_THREAD; ++ii ) {
      sum += q[ii].x * kv[ii].x; 
      sum += q[ii].y * kv[ii].y; 
      sum += q[ii].z * kv[ii].z; 
      sum += q[ii].w * kv[ii].w; 
    }

    for( int ii = 0; ii < FLOAT4s_PER_THREAD; ++ii ) {
      kv[ii].x *= gamma; 
      kv[ii].y *= gamma; 
      kv[ii].z *= gamma;
      kv[ii].w *= gamma;
    }

    // Finalize the computation of the sum (if we have more than 1 thread per head).
    if( THREADS_PER_HEAD > 1 ) {

      // Finalize the sum for each head.
      #pragma unroll
      for( int mask = THREADS_PER_HEAD / 2; mask >= 1; mask /= 2 ) {
        sum += __shfl_xor_sync(uint32_t(-1), sum, mask);
      }

      // Store to shared memory.
      if( vo < M && vi == 0 ) {
        smem_o[smem_curr*smem_buffer_elts + vo] = sum ;
      }

      // Make sure the data is in shared memory.
      __syncthreads();

      // Active threads read the data to store.
      if( active_v ) {
        sum = smem_o[smem_curr*smem_buffer_elts + tidx];
      }

    } // THREADS_PER_HEAD > 1.

    // Store the output. All the threads are active.
    if( active_v ) {
      *out_ptr = sum;
    }

    // Move to next location.
    out_ptr += params.o_stride_L;
    

    // Move the shared memory buffer.
    smem_curr = (smem_curr + 1) % 2;

    // Store to shared memory for Q and K.
    if( !is_last && tidx < E ) {
      smem_q[smem_curr*smem_buffer_elts + tidx] = ldg_q;
      smem_k[smem_curr*smem_buffer_elts + tidx] = ldg_k;
    }

    // Store to shared memory for V.
    if( !is_last && tidx < M ) {
      smem_v[smem_curr*smem_buffer_elts + tidx] = ldg_v;
    }
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template< int E, int THREADS_PER_HEAD>
int lmha_(const Lmha_params<float> &params) {
  // The M dimension rounded up to 4.
  int M = round_up(params.M, 4);

  // The number of threads in the block.
  int block = round_up(max(E, M*THREADS_PER_HEAD), 32);
  if( block > 512 || params.B > 65535 ) {
    return 1;
  }

  // Prepare the kernel.
  dim3 grid(params.H, params.B);
  size_t smem = smem_buffer_elts_<E>(params)*2*sizeof(float);
  lmha_kernel<E, THREADS_PER_HEAD><<<grid, block, smem>>>(params);
  return 0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

int lmha(const Lmha_params<float> &params) {
  int blocks = params.B * params.H;
  int res = 1;
  if( blocks < LOW_OCCUPANCY_THRESHOLD ) { 
           if( params.E <=  32 ) {
      res = lmha_low_occupancy_< 32 >(params, blocks);
    } else if( params.E <=  64 ) {
      res = lmha_low_occupancy_< 64 >(params, blocks);
    } else if( params.E <= 128 ) {
      res = lmha_low_occupancy_<128>(params, blocks);
    } else if( params.E <= 256 ) {
      res = lmha_low_occupancy_<256>(params, blocks);
    }
  } else {
           if( params.E <=  32 ) {
      res = lmha_< 32, 1>(params);
    } else if( params.E <=  48 ) {
      res = lmha_< 48, 1>(params);
    } else if( params.E <=  64 ) {
      res = lmha_< 64, 1>(params);
    } else if( params.E <= 128 ) {
      res = lmha_<128, 2>(params);
    } else if( params.E <= 256 ) {
      res = lmha_<256, 4>(params);
    }
  }
  return res;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template< typename T >
inline void set_params(Lmha_params<T> &params,
                       const torch::Tensor q,
                       const torch::Tensor k,
                       const torch::Tensor v,
                       const torch::Tensor gamma,
                       torch::Tensor       o) {

  // Define the pointers.
  params.out = o.data_ptr<T>();
  params.q   = q.data_ptr<T>();
  params.k   = k.data_ptr<T>();
  params.v   = v.data_ptr<T>();
  params.gamma = gamma.data_ptr<T>();

  // Define the strides.
  params.q_stride_B = (int) q.stride(0);
  params.q_stride_H = (int) q.stride(1);
  params.q_stride_L = (int) q.stride(2);
  params.k_stride_B = (int) k.stride(0);
  params.k_stride_H = (int) k.stride(1);
  params.k_stride_L = (int) k.stride(2);
  params.v_stride_B = (int) v.stride(0);
  params.v_stride_H = (int) v.stride(1);
  params.v_stride_L = (int) v.stride(2);
  params.o_stride_B = (int) o.stride(0);
  params.o_stride_H = (int) o.stride(1);
  params.o_stride_L = (int) o.stride(2);

  // Extract the dimensions.
  int N = q.size(0);
  int H = q.size(1);
  int L = q.size(2);
  int E = q.size(3);
  int M = v.size(3);

  params.B = N;
  params.L = L;
  params.H  = H;
  params.E = E;
  params.M = M;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

int lmha_fwd(const torch::Tensor queries,
             const torch::Tensor keys,
             const torch::Tensor values,
             const torch::Tensor gamma,
             torch::Tensor product) {

  // Make sure that we are using the correct GPU device
  torch::DeviceGuard _guard(queries.device());
  // Make sure the inner-most dimension of the tensors is packed.
  assert(queries.stride(3) == 1);
  assert(keys   .stride(3) == 1);
  assert(values .stride(3) == 1);
  assert(product.stride(3) == 1);

  // Extract the dimensions.
  int N = queries.size(0);
  int H = queries.size(1);
  int L = queries.size(2);
  int E = queries.size(3);
  int M = values.size (3);

  // The structure of params.
  Lmha_params<float> params;
  set_params(params, queries, keys, values, gamma, product);

  // Launch the kernel.
  return lmha(params);
}
}

////////////////////////////////////////////////////////////////////////////////////////////////////

typedef torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits> float_accessor;
typedef torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> gamma_accessor;

typedef torch::PackedTensorAccessor64<float, 4, torch::RestrictPtrTraits> float_accessor64;
typedef torch::PackedTensorAccessor64<float, 1, torch::RestrictPtrTraits> gamma_accessor64;

#define E_BLOCK_SIZE 8

__global__ void causal_dot_product_kernel(
    const float_accessor queries,
    const float_accessor keys,
    const float_accessor values,
    const gamma_accessor gamma,
    float_accessor result,
    const int N,
    const int H,
    const int L,
    const int E,
    const int M
) {
    int n = blockIdx.y;
    int h = blockIdx.z;

    int e_start = blockIdx.x * E_BLOCK_SIZE;
    int m = threadIdx.x % M;

    extern __shared__ float shared_mem[];
    float* shared_kv = shared_mem;

    for (int e_local = 0; e_local < E_BLOCK_SIZE && e_local + e_start < E; e_local++) {
      shared_kv[m + e_local * M] = 0;
    }

    for (int t=0; t<L; t++) {
      float res = 0;
      for (int e_local = 0; e_local < E_BLOCK_SIZE && e_local + e_start < E; e_local++) {
        // shared_kv[e_local*M + m] *= 0.5
        shared_kv[e_local*M + m] += keys[n][h][t][e_local + e_start] * values[n][h][t][m];
        res += queries[n][h][t][e_local + e_start] * shared_kv[e_local*M + m];
        shared_kv[e_local*M + m] *= gamma[h];
      }
      atomicAdd(
          &result[n][h][t][m],
          res
      );
    }
}
#endif
////////////////////////////////////////////////////////////////////////////////////////////////////

void causal_dot_product_(const torch::Tensor queries,
                         const torch::Tensor keys,
                         const torch::Tensor values,
                         const torch::Tensor gamma,
                         torch::Tensor product) {
    // Make sure that we are using the correct GPU device
    torch::DeviceGuard _guard(queries.device());

    int N = queries.size(0);
    int H = queries.size(1);
    int L = queries.size(2);
    int E = queries.size(3);
    int M = values.size(3);

    const int blocks_per_sequence = (E + E_BLOCK_SIZE - 1) / E_BLOCK_SIZE;

    dim3 blockDim(M, 1, 1);
    dim3 gridDim(blocks_per_sequence, N, H);
    const int shared_mem_forward = E_BLOCK_SIZE * M * sizeof(float);

    causal_dot_product_kernel<<<gridDim, blockDim, shared_mem_forward>>>(
      queries.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
      keys.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
      values.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
      gamma.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
      product.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
      N, H, L, E, M
    );
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void causal_dot_product(const torch::Tensor queries,
                        const torch::Tensor keys,
                        const torch::Tensor values,
                        const torch::Tensor gamma,
                        torch::Tensor product) 
{
  #ifdef ENABLE_NVIDIA_OPTIMIZATIONS
  int fallback = nvidia::lmha_fwd(queries, keys, values, gamma, product);
  // printf("fallback:%d\n",fallback);
  #else
    int fallback = 1;
  #endif
    if( fallback ) {
      causal_dot_product_(queries, keys, values, gamma, product);
    }
}


////////////////////////////////////////////////////////////////////////////////////////////////////

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def(
        "causal_dot_product",
        &causal_dot_product,
        "Compute the weighted sum of values but attending only to previous "
        "values."
    );
}
